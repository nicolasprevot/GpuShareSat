/***************************************************************************************
 GpuShareSat -- Copyright (c) 2020, Nicolas Prevot

Permission is hereby granted, free of charge, to any person obtaining a copy of this software and
associated documentation files (the "Software"), to deal in the Software without restriction,
including without limitation the rights to use, copy, modify, merge, publish, distribute,
sublicense, and/or sell copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all copies or
substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT
NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,
DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT
OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 **************************************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <thread>
#include <csignal>

#include "Helper.cuh"
#include "GpuMultiSolver.cuh"
#include "satUtils/Dimacs.h"
#include "CompositionRoot.cuh"
#include "core/Solver.h"
#include "satUtils/InitHelper.h"
#include "utils/System.h"

using namespace Glucose;

Finisher finisher;

void SIGINT_exit(int signum) {
    finisher.cancel();
}


int runGpuSolver(CompositionRoot &compRoot, GpuOptions &gpuOptions, CommonOptions &commonOpts, double memUsedOneSolver) {
    GpuMultiSolver& msolver = *compRoot.gpuMultiSolver;
    msolver.setVerbosity(commonOpts.getVerbosity());
    lbool ret = l_Undef;
    if (commonOpts.doPreprocessing()) {
        ret = msolver.simplify();
    }
    if (ret == l_Undef) {
        // We have an approximation of the memory used for one solver. We don't take into account the memory used for the gpu itself
        // or other things.
        int cpuSolverCount = gpuOptions.getNumberOfCpuThreads(msolver.getVerbosity().global, memUsedOneSolver);
        int warpsPerBlock = compRoot.gpuDims.threadsPerBlock / WARP_SIZE;
        compRoot.hostAssigs->growSolverAssigs(cpuSolverCount, warpsPerBlock, compRoot.gpuDims.blockCount * warpsPerBlock);
        ret = msolver.solve(cpuSolverCount);
    }
    printResult(ret);

    if (msolver.getVerbosity().showModel && ret==l_True) {
        printModel(stdout, msolver.getModel());
    }
    return getReturnCode(ret);
}

int main(int argc, char **argv)
{
    // It can be necessary for debugging if we print a lot
    // 100 Megs
    // hipDeviceSetLimit(hipLimitPrintfFifoSize, 1048576 * 100);

    printf("c\nc This is glucose-gpu 1.0 --  based on MiniSAT (Many thanks to MiniSAT team)\nc\n");
    signal(SIGINT, SIGINT_exit);
    signal(SIGXCPU,SIGINT_exit);
    try {
        CommonOptions commonOptions;
        GpuOptions gpuOptions;
        setUsageHelp("c USAGE: %s [options] <input-file> <result-output-file>\n\n  where input may be either in plain or gzipped DIMACS.\n");
        parseOptions(argc, argv, true);
        commonOptions.applyTimeAndCpuLim();
        TimePrinter timePrinter("taken total");

        gzFile in = getInputFile(argc, argv);

        DimacsParser parser(in);

        // Note: seems this flag has to be set before a stream is created (rather than on the thread using the stream)
        // The reason for setting it here instead of composition root is that:
        // If set in comp root, it would be set for some unit tests but not other
        // but this flag needs to be set before the device starts to run, so it wouldn't work
        // this flag is good if there are few threads
        // exitIfError(hipSetDeviceFlags(hipDeviceScheduleBlockingSync), POSITION);
        CompositionRoot compRoot(gpuOptions, finisher, parser.nVars());

        // Note: cuda uses a gigantic amount (gigabytes) of virtual memory that is almost never used, to get the whole physical memory into virtual
        // memory. We'd like to not count that in the memory limits. But not easy to get just this one

        // Unfortunately, memUsed here includes some memory that is really used, but it's not easy to tell what is
        commonOptions.applyMemLim(memUsed());
        double memUsedBeforeClauses = memUsed();
        parser.fillClauses(*compRoot.gpuMultiSolver);
        double memUsedAfterClauses = memUsed();
        gzclose(in);

        // We only look at differentials of memory to get the memory per solver
        return runGpuSolver(compRoot, gpuOptions, commonOptions, memUsedAfterClauses - memUsedBeforeClauses
                + compRoot.gpuMultiSolver->getMemUsedCreateOneSolver());
    } catch (OutOfMemoryException&){
        printf("c ===================================================================================================\n");
        printf("INDETERMINATE. OutOfMemoryException\n");
        exit(0);
    }
}
