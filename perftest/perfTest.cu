/***************************************************************************************
 GpuShareSat -- Copyright (c) 2020, Nicolas Prevot

Permission is hereby granted, free of charge, to any person obtaining a copy of this software and
associated documentation files (the "Software"), to deal in the Software without restriction,
including without limitation the rights to use, copy, modify, merge, publish, distribute,
sublicense, and/or sell copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all copies or
substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT
NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,
DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT
OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 **************************************************************************************************/
#define BOOST_TEST_MODULE perftest_module
#include <boost/test/unit_test.hpp>
#include "gpu/Helper.cuh"
#include "gpu/Assigs.cuh"
#include "gpu/Clauses.cuh"
#include "gpu/GpuHelpedSolver.h"
#include "gpu/GpuRunner.cuh"
#include "satUtils/SolverTypes.h"
#include "core/Solver.h"
#include "gpuShareLib/Utils.h"
#include <hip/hip_runtime.h>
#include <mtl/Vec.h>
#include <algorithm>
#include <cstdlib>
#include <ctime>
#include <memory>
#include "gpu/GpuRunner.cuh"
#include "testUtils/TestHelper.cuh"
#include "gpuShareLib/Utils.h"
#include "gpu/my_make_unique.h"
#include "utils/Utils.h"

namespace Glucose {

int getDiffMicros(timespec begin, timespec end) {
    return (end.tv_sec - begin.tv_sec) * 1000000 + (end.tv_nsec - begin.tv_nsec) / 1000;
}

std::unique_ptr<GpuOptions> getOptions(int clCount, int clMinSize, int clMaxSize) {
    auto ptr = my_make_unique<GpuOptions>();
    ptr -> blockCount = 10;
#ifndef NDEBUG
    ptr -> threadsPerBlock = 150;
#else
    ptr -> threadsPerBlock = 1024;
#endif
    // make sure that we don't reduce the db
    ptr -> gpuFirstReduceDb = 1e9;
    return ptr;
}


class PerfFixture : public GpuFixture {
public:
    int clauseCount;
    int clMinSize;
    int clMaxSize;
    PerfFixture(int clCount = 1000000, int clMinSize = 12, int clMaxSize = 20, int varCount = 500, int solverCount = 1);
};

void maybeSetVariable(double &seed, GpuHelpedSolver &solver, int var) {
    int p = irand(seed, 3);
    if (p == 0 || p == 1) {
        solver.newDecisionLevel();
        solver.uncheckedEnqueue(mkLit(var, p == 1));
    }
}

void resetAllVariables(double &seed, GpuHelpedSolver &solver) {
    solver.cancelUntil(0);
    for (int i = 0; i < solver.nVars(); i++) {
        maybeSetVariable(seed, solver, i);
    }
}

// This has to be set before the gpu starts, so at the beginning of each test
void setDeviceFlags() {
    unsigned int flags;
    hipGetDeviceFlags(&flags);
    if (flags & hipDeviceScheduleBlockingSync == 0) {
        exitIfError(hipSetDeviceFlags(hipDeviceScheduleBlockingSync), POSITION);
    }
}

PerfFixture::PerfFixture(int _clauseCount, int _clMinSize, int _clMaxSize, int nVars, int solverCount) :
    clauseCount(_clauseCount),
    clMinSize(_clMinSize),
    clMaxSize(_clMaxSize),
    GpuFixture(*(getOptions(_clauseCount, _clMinSize, _clMaxSize)), nVars, solverCount, 2000) {
    srand(25);
    vec<Lit> lits;
    ContigCopier cc(true);
    hipStream_t &stream = gpuClauseSharer.sp.get();
    GpuDims gpuDims {10, 256};
    double seed = 0.4;
    for (int cl = 0; cl < clauseCount; cl++) {
        lits.clear();
        int size = irand(seed, clMinSize, clMaxSize);
        for (int l = 0; l < size; l++) {
            lits.push(randomLit(seed, nVars));
        }
        gpuClauseSharer.clauses->addClause(MinHArr<Lit>(lits.size(), &lits[0]), 5);
        // HClauses is designed to copy clauses in small chunks, not a large amount at once
        if (cl % 5000 == 0) {
            copyToDeviceAsync(*gpuClauseSharer.clauses, stream, cc, gpuDims);
            exitIfError(hipStreamSynchronize(stream), POSITION);
        }
    }
    copyToDeviceAsync(*gpuClauseSharer.clauses, stream, cc, gpuDims);
    exitIfError(hipStreamSynchronize(stream), POSITION);
}

// print all the wrong clauses
BOOST_AUTO_TEST_CASE(testPrintClauses) {
    setDeviceFlags();
    PerfFixture fx(300000, 10, 11);
    double seed = 0.6;
    resetAllVariables(seed, *(fx.solvers[0]));
    fx.solvers[0]->tryCopyTrailForGpu(fx.solvers[0]->decisionLevel());
    execute(fx.gpuClauseSharer);
    Lit array[MAX_CL_SIZE];
    GpuClauseId gpuClauseId;
    MinHArr<Lit> lits;

    while (fx.gpuClauseSharer.reported->popReportedClause(0, lits, gpuClauseId)) {
        // vec doesn't have a sort method, so let's use an array instead
        for (int j = 0; j < lits.size(); j++) {
            array[j] = lits[j];
        }
        std::sort(array, array + lits.size());
        printf(">> ");
        for (int j = 0; j < lits.size(); j++) {
            PRINTV(array[j]);
        }
        printf("\n");
    }
}

BOOST_AUTO_TEST_CASE(testPerf) {
    setDeviceFlags();
    PerfFixture fx(2000000, 12, 20, 500, 1);

    exitIfLastError(POSITION);
    timespec begin, gpuDone, end;
    long gpuExecTimeMicros = 0;
    long importTimeMicros = 0;
    exitIfLastError(POSITION);
    // having n = 2000 is really to slow if we're in debug
    // But in release, to have a consistent result, we need a big enough
    // value for n
#ifdef NDEBUG
    long n = 2000;
#else
    long n = 15;
#endif

    double seed = 0.6;
    printf("solver count: %d\n", fx.solvers.size());
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < fx.solvers.size(); j++) {
            resetAllVariables(seed, *(fx.solvers[j]));
            fx.solvers[j]->tryCopyTrailForGpu(fx.solvers[j]->decisionLevel());
        }
        clock_gettime(CLOCK_REALTIME, &begin);
        execute(fx.gpuClauseSharer);
        clock_gettime(CLOCK_REALTIME, &gpuDone);
        // This is partly because we can't add more assignments unless we read clauses for existing assignments
        bool a;
        for (int j = 0; j < fx.solvers.size(); j++) fx.solvers[j]->gpuImportClauses(a);
        exitIfLastError(POSITION);
        clock_gettime(CLOCK_REALTIME, &end);
        gpuExecTimeMicros += getDiffMicros(begin, gpuDone);
        importTimeMicros += getDiffMicros(gpuDone, end);
    }

    if (gpuExecTimeMicros + importTimeMicros == 0) {
        printf("no time passed");
    }
    else {
        printf("gpu exec time taken: %ld micros\n", gpuExecTimeMicros);
        printf("import time taken: %ld micros\n", importTimeMicros);
        printf("wrong clause count: %ld\n", fx.gpuClauseSharer.getGlobalStat(gpuReports));
        printf("clause count: %d\n", fx.clauseCount);
        printf("executions per seconds: %ld\n", (n * 1000000)/ (gpuExecTimeMicros + importTimeMicros));
        printf("reads per microsecond: %ld\n", n * fx.clauseCount * (1 + (fx.clMinSize + fx.clMaxSize) / 2) / (gpuExecTimeMicros));
    }
#ifdef NDEBUG
    BOOST_CHECK_EQUAL(19739, fx.gpuClauseSharer.getGlobalStat(gpuReports));
#else
    BOOST_CHECK_EQUAL(143, fx.gpuClauseSharer.getGlobalStat(gpuReports));
#endif
    exitIfLastError(POSITION);
}

BOOST_AUTO_TEST_CASE(testReportedAreValid) {
    setDeviceFlags();
    PerfFixture fx(1000000, 10, 11, 500);
    GpuHelpedSolver &solver = *(fx.solvers[0]);
    exitIfLastError(POSITION);
    bool foundEmptyClause = false;
    int importedValidLastTime = 0;
    int importedLastTime = 0;
    double seed = 0.8;
    resetAllVariables(seed, *(fx.solvers[0]));
    // If the gpu reports some clauses: at least one of them must be valid
    // Because the cpu solver then changes its variables because of this one,
    // the next clauses may not be valid
    while (true) {
        fx.solvers[0]->tryCopyTrailForGpu(fx.solvers[0]->decisionLevel());
        // the first maybExecute will only start the run but not get the results, so execute twice
        execute(fx.gpuClauseSharer);
        CRef conflict = solver.gpuImportClauses(foundEmptyClause);
        int reported = solver.stats[nbImported], importedValid = solver.stats[nbImportedValid];
        printf("%d clauses imported out of which %d valid\n", reported, importedValid);

        vec<Lit> clauseLits;

        // continue as long as we get some clauses
        if (solver.stats[nbImported] == importedLastTime) {
            break;
        }
        importedLastTime = solver.stats[nbImported];
        ASSERT_OP(solver.stats[nbImportedValid], >, importedValidLastTime);
        importedValidLastTime = solver.stats[nbImportedValid];

        // If the solver got a conflict at level n, it's still at level n.
        // We need to cancel it until the previous level because otherwise, it will get the same conflict over and over
        if (conflict != CRef_Undef) {
            if (solver.decisionLevel() == 0) break;
            solver.cancelUntil(solver.decisionLevel() - 1);
        }
    }
    exitIfError(hipStreamSynchronize(fx.gpuClauseSharer.sp.get()), POSITION);
    exitIfLastError(POSITION);
}

}
