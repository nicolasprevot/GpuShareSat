#include "hip/hip_runtime.h"
/***************************************************************************************
 GpuShareSat -- Copyright (c) 2020, Nicolas Prevot

Permission is hereby granted, free of charge, to any person obtaining a copy of this software and
associated documentation files (the "Software"), to deal in the Software without restriction,
including without limitation the rights to use, copy, modify, merge, publish, distribute,
sublicense, and/or sell copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all copies or
substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT
NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,
DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT
OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 **************************************************************************************************/
#include "Helper.cuh"
#include "GpuRunner.cuh"
#include "Clauses.cuh"
#include "Reported.cuh"
#include "Assigs.cuh"
#include "Reporter.cuh"
#include "gpuShareLib/Utils.h"
#include "CorrespArr.cuh"
#include "my_make_unique.h"
#include <thread>         // std::this_thread::sleep_for
#include "GpuClauseSharer.h"

// #define PRINTCN_ALOT 1

namespace GpuShare {
struct ReportComputer {
    // if all literals seen so far are false in the current clause
    Vals allFalse;
    // all are false, exect for one which is undefined
    Vals justOneUndefined;

    __device__ void init(Vals startVal) {
        justOneUndefined = 0;
        allFalse = startVal;
    }

    __device__ bool nothingToReport() {
        return !(allFalse | justOneUndefined);
    }

    __device__ void update(Vals canBeFalse, Vals canBeUndef) {
        justOneUndefined = (allFalse & canBeUndef) | (justOneUndefined & canBeFalse);
        allFalse = allFalse & canBeFalse;
    }

    __device__ Vals getToReport() {
        return allFalse | justOneUndefined;
    }
};

__global__ void init(DValsPerId<VarUpdate> varUpdates, DArr<DOneSolverAssigs> dOneSolverAssigs, DAssigAggregates aggregates, DReporter<ReportedClause> dReporter, DValsPerId<AggCorresp> aggCorresps) {
    dReporter.clear();
    dUpdateAssigs(varUpdates, dOneSolverAssigs, aggCorresps, aggregates); 
}

__global__ void initClauses(DClauseUpdates dClauseUpdates, DClauses dClauses) {
    updateClauses(dClauseUpdates, dClauses);
}

__device__ void dCheckOneClauseOneSolver(DOneSolverAssigs dOneSolverAssigs, DAssigAggregates dAssigAggregates,
        Lit* startLitPt, Lit* endLitPt, DReporter<ReportedClause> dReporter, int solverId, GpuCref gpuCref) {
    Lit* litPt = startLitPt;
    ReportComputer reportComputer;
    reportComputer.init(dOneSolverAssigs.startVals);
    while (litPt < endLitPt) {
        Lit lit = *litPt;
        Vals va = dVar(lit);
        Vals isFalse = dOneSolverAssigs.multiLBools[va].isTrue;
        if (!dSign(lit)) {
            isFalse = ~ isFalse;
        }
        Vals def = dOneSolverAssigs.multiLBools[va].isDef;
        reportComputer.update(isFalse & def, ~def);
        if (reportComputer.nothingToReport()) {
            return;
        }
        litPt += WARP_SIZE;
    }
    ASSERT_OP_C(gpuCref.clSize, >=, 1);
    dReporter.report(ReportedClause {reportComputer.getToReport(), solverId, gpuCref}, getThreadId());
}

// note: this method doesn't unset the first bit, unlike the cpu one
__device__ void dGetFirstBitPosFast(Vals &val, int &pos) {
    if ((val & 0xFFFF) == 0) {
        val = val >> 16;
        pos += 16;
    }
    if ((val & 0xFF) == 0) {
        val = val >> 8;
        pos += 8;
    }
    if ((val & 0xF) == 0) {
        val = val >> 4;
        pos += 4;
    }
    if ((val & 0x3) == 0) {
        val = val >> 2;
        pos += 2;
    }
    if ((val & 0x1) == 0) {
        val = val >> 1;
        pos += 1;
    }
    assert(val & 1);
}

__device__ void dCheckOneClauseAllSolvers(DArr<DOneSolverAssigs> dOneSolverAssigs, DAssigAggregates dAssigAggregates,
        Lit* startLitPt, Lit*endLitPt, DReporter<ReportedClause> dReporter, GpuCref gpuCref, Vals bits, long &clauseTestsOnAssigs) {
    int pos = 0;
    while (true) {
        if (bits == 0) {
            return;
        }
        dGetFirstBitPosFast(bits, pos);
        int solver = dAssigAggregates.getSolver(pos);
        dCheckOneClauseOneSolver(dOneSolverAssigs[solver], dAssigAggregates, startLitPt, endLitPt, dReporter, solver, gpuCref);
        int newPos = dAssigAggregates.getEndBitPos(solver);
        bits = bits >> (newPos - pos);
        pos = newPos;
        clauseTestsOnAssigs++;
    }
}

// This method is performance critical. So it's dealing with pointers directly, which isn't super safe
__global__ void dFindClauses(DArr<DOneSolverAssigs> dOneSolverAssigs, DAssigAggregates dAssigAggregates,
        DClauses dClauses, DReporter<ReportedClause> dreporter, DArr<long> oneSolverCheckArr) {
    int clSize, clIdStart, clIdEnd;
    int threadId = getThreadId();
    ReportComputer reportComputer;
    dClauses.getClsForThread(threadId, clSize, clIdStart, clIdEnd);
    ASSERT_OP_C(clSize, >=, 1);
    for (int clId = clIdStart; clId < clIdEnd; clId += WARP_SIZE) {
        Lit *startLitPt = dClauses.getStartAddrForClause(clSize, clId);
        Lit *litPt = startLitPt;
        Lit *endLitPt = litPt + WARP_SIZE * clSize;

        reportComputer.init(dAssigAggregates.startVals);
        GpuCref gpuCref {clSize, clId};
        while (litPt < endLitPt) {
#ifndef NDEBUG
            dClauses.assertInSize(clSize, litPt);
#endif
            Lit lit = *litPt;

            Vals va = dVar(lit);
            ASSERT_OP_MSG_C(va, <, dAssigAggregates.multiAggs.size(), PRINTCN(lit); PRINTCN(clId); PRINTCN(clSize); PRINTCN(dClauses.getClCount(clSize)));
            MultiAgg &multiAgg = dAssigAggregates.multiAggs[va];
            assert((~ (multiAgg.canBeTrue | multiAgg.canBeFalse | multiAgg.canBeUndef)) == 0);
            Vals val;
            if (dSign(lit)) {
                val = multiAgg.canBeTrue;
            }
            else {
                val = multiAgg.canBeFalse;
            }
            reportComputer.update(val, multiAgg.canBeUndef);
            if (reportComputer.nothingToReport()) {
                // gotos are bad, but this code is performance critical so it's worth it
                goto next;
            }
            litPt += WARP_SIZE;
        }
        dCheckOneClauseAllSolvers(dOneSolverAssigs, dAssigAggregates,
            startLitPt, endLitPt, dreporter, gpuCref, reportComputer.getToReport(), oneSolverCheckArr[threadId]);
next: ;
    }
}

GpuRunner::GpuRunner(HostClauses &_hostClauses, HostAssigs &_hostAssigs, Reported &_reported, GpuDims gpuDimsGuideline, bool _quickProf,
        int _countPerCategory, hipStream_t &_stream, std::vector<unsigned long> &_globalStats, const Logger &logger) :
    warpsPerBlock(gpuDimsGuideline.threadsPerBlock / WARP_SIZE),
    blockCount(gpuDimsGuideline.blockCount),
    hasRunOutOfGpuMemoryOnce(false),
    lastInAssigIdsPerSolver(1),
    clauseTestsOnAssigs(false, false, logger),
    quickProf(_quickProf),
    hostAssigs(_hostAssigs),
    hostClauses(_hostClauses),
    reported(_reported),
    categoryCount(gpuDimsGuideline.blockCount),
    countPerCategory(_countPerCategory), 
    cpuToGpuContigCopier(logger, true),
    gpuToCpuContigCopier(logger, true),
    stream(_stream),
    globalStats(_globalStats) {

}

void GpuRunner::prepareOneSolverChecksAsync(int threadCount, hipStream_t &stream) {
    int oldSize = clauseTestsOnAssigs.size();
    // gpuThreadCount can change with every run
    if (oldSize < threadCount) {
        clauseTestsOnAssigs.resize(threadCount, false);
        for (int i = oldSize; i < threadCount; i++) {
            clauseTestsOnAssigs[i] = 0;
        }
        clauseTestsOnAssigs.copyAsync(hipMemcpyHostToDevice, stream, oldSize, threadCount);
    }
}

void GpuRunner::wholeRun(bool canStart) {
    // make sure that we've at least finished the previous copy from cpu to gpu
    exitIfError(hipEventSynchronize(cpuToGpuCopyDone.get()), POSITION);
    // The gpu is currently processing some assignments. Let's start preparing the next assignments once findClauses is done. At that point, the gpu will still
    // have to set all assigs to last and to copy things back to the cpu
    if (prevReporter) {
        hipEventSynchronize(afterFindClauses.get());
        if (quickProf) {
            float ms;
            exitIfError(hipEventElapsedTime(&ms, beforeFindClauses.get(), afterFindClauses.get()), POSITION);
            globalStats[timeSpentTestingClauses] += ms * 1000;
        }
    }
    int nextInAssigIdsPerSolver = -1;
    std::unique_ptr<Reporter<ReportedClause>> nextReporter;
    bool startingNew = false;
    bool notEnoughGpuMemory = false;
    if (canStart) {
        nextInAssigIdsPerSolver = (lastInAssigIdsPerSolver + 1) % 2;
        startGpuRunAsync(stream, assigIdsPerSolver[nextInAssigIdsPerSolver], nextReporter, startingNew, notEnoughGpuMemory);
    }
    if (prevReporter) {
        gatherGpuRunResults(assigIdsPerSolver[lastInAssigIdsPerSolver], *prevReporter);
    }
    if (startingNew) {
        scheduleGpuToCpuCopyAsync(stream);
        lastInAssigIdsPerSolver = nextInAssigIdsPerSolver;
        prevReporter.swap(nextReporter);
    } else {
        prevReporter.reset();
    }
    if (notEnoughGpuMemory) {
        hostClauses.reduceDb(stream);
        hasRunOutOfGpuMemoryOnce = true;
    }
}

/*
struct InitParams {
    DArr<DArr<VarUpdate>> varUpdates;
    DArr<DOneSolverAssigs> dOneSolverAssigs;
    DAssigAggregates aggregates;
    DReporter<ReportedClause> dReporter;
    DClauseUpdates dClauseUpdates;
    DClauses dClauses;
}

*/
void GpuRunner::startGpuRunAsync(hipStream_t &stream, std::vector<AssigIdsPerSolver> &assigIdsPerSolver, std::unique_ptr<Reporter<ReportedClause>> &reporter, bool &started, bool &notEnoughGpuMemory) {
#ifdef PRINTCN_ALOT
    printf("startGpuRunAsync\n");
#endif

    // note that there could still be some updates from the previous run that still need to be read from the gpu to set all vars to last
    // it is fine to clear here though, since the copy will be enqueued on the same stream as the one where all vars will be set to last
    // so it will happen after
    cpuToGpuContigCopier.clear(false);

    ClUpdateSet clUpdateSet = hostClauses.getUpdatesForDevice(stream, cpuToGpuContigCopier);
    // globalStats[gpuClauses] at this point includes clauses that are about to be copied to the device
    if (globalStats[gpuClauses] == 0) {
        started = false;
        notEnoughGpuMemory = false;
        return;
    }
    RunInfo runInfo = hostClauses.makeRunInfo(stream, cpuToGpuContigCopier);

    if (!runInfo.succeeded()) {
        // Failed to allocate the memory 
        // it's fine not to call initClauses since the next thing we'll do is reduceDb which will
        // sync device and host anyway
        started = false;
        notEnoughGpuMemory = true;
        return;
    }

    TimeGauge tg(globalStats[timeSpentFillingAssigs], quickProf);
    AssigsAndUpdates assigsAndUpdates = hostAssigs.fillAssigsAsync(cpuToGpuContigCopier, assigIdsPerSolver, stream);
    tg.complete();

    if (!cpuToGpuContigCopier.tryCopyAsync(hipMemcpyHostToDevice, stream)) {
        THROW();
    }
    exitIfError(hipEventRecord(cpuToGpuCopyDone.get(), stream), POSITION);

    gpuToCpuContigCopier.clear(false);
    reporter = my_make_unique<Reporter<ReportedClause>>(gpuToCpuContigCopier, stream, countPerCategory, categoryCount);
    auto dReporter = reporter->getDReporter();
    DClauses dClauses = runInfo.getDClauses();

    ASSERT_OP_C(warpsPerBlock, >, 0);

    runGpuAdjustingDims(warpsPerBlock, warpsPerBlock * blockCount, [&] (int blockCount, int threadsPerBlock) {
        init<<<blockCount, threadsPerBlock, 0, stream>>>(assigsAndUpdates.dAssigUpdates.get(), assigsAndUpdates.assigSet.dSolverAssigs.getDArr(), assigsAndUpdates.assigSet.dAssigAggregates, dReporter, assigsAndUpdates.assigSet.aggCorresps.get());
    });
    exitIfError(hipGetLastError(), POSITION);

    runGpuAdjustingDims(warpsPerBlock, warpsPerBlock * blockCount, [&] (int blockCount, int threadsPerBlock) {
        initClauses<<<blockCount, threadsPerBlock, 0, stream>>>(clUpdateSet.getDClauseUpdates(), dClauses);
    });

    prepareOneSolverChecksAsync(runInfo.warpCount * WARP_SIZE, stream);
    if (quickProf) exitIfError(hipEventRecord(beforeFindClauses.get(), stream), POSITION);

    // Only this run uses runInfo.warpCount for the dimensions
    runGpuAdjustingDims(warpsPerBlock, runInfo.warpCount, [&] (int blockCount, int threadsPerBlock) {
        dFindClauses<<<blockCount, threadsPerBlock, 0, stream>>>(assigsAndUpdates.assigSet.dSolverAssigs.getDArr(),
            assigsAndUpdates.assigSet.dAssigAggregates, dClauses, dReporter, clauseTestsOnAssigs.getDArr());
    });
    exitIfError(hipEventRecord(afterFindClauses.get(), stream), POSITION);
    setAllAssigsToLastAsync(warpsPerBlock, warpsPerBlock * blockCount, assigsAndUpdates, stream);
    started = true;
    notEnoughGpuMemory = false;
}

void GpuRunner::scheduleGpuToCpuCopyAsync(hipStream_t &stream) {
    exitIfFalse(gpuToCpuContigCopier.tryCopyAsync(hipMemcpyDeviceToHost, stream), POSITION);
    exitIfError(hipEventRecord(gpuToCpuCopyDone.get(), stream), POSITION);
}

int getTotalAssigCount(std::vector<AssigIdsPerSolver> &assigIdsPerSolver) {
    int res = 0;
    for (int i = 0; i < assigIdsPerSolver.size(); i++) {
        res += assigIdsPerSolver[i].assigCount;
    }
    return res;
}

void GpuRunner::gatherGpuRunResults(std::vector<AssigIdsPerSolver> &assigIdsPerSolver, Reporter<ReportedClause> &reporter) {
    globalStats[gpuRuns]++;
    exitIfError(hipEventSynchronize(gpuToCpuCopyDone.get()), POSITION);
    if (reporter.getCopiedToHost(reportedCls)) {
        countPerCategory *= 2;
    }

    int assigsCount = getTotalAssigCount(assigIdsPerSolver);
    int clCount = globalStats[gpuClauses];
    globalStats[totalAssigClauseTested] += clCount * assigsCount;
    globalStats[clauseTestsOnGroups] += clCount;
    globalStats[gpuReports] += reportedCls.size();
#if PRINTCN_ALOT == 1
    printf("filling reported with %d assigs and %d clauses\n", assigsCount, reportedCls.size());
#endif
    for (int i = 0; i < reportedCls.size(); i++) {
        ASSERT_OP_C(reportedCls[i].gpuCref.clSize, >=, 1);
        hostClauses.bumpClauseActivity(reportedCls[i].gpuCref);
    }
    {
        TimeGauge tg(globalStats[timeSpentFillingReported], quickProf);
        reported.fill(assigIdsPerSolver, reportedCls);
    }
}

long GpuRunner::getClauseTestsOnAssigs() {
    clauseTestsOnAssigs.copyAsync(hipMemcpyDeviceToHost, stream);
    exitIfError(hipStreamSynchronize(stream), POSITION);
    return getSum(clauseTestsOnAssigs);
}

}
