#include "hip/hip_runtime.h"
/***************************************************************************************
 GpuShareSat -- Copyright (c) 2020, Nicolas Prevot

Permission is hereby granted, free of charge, to any person obtaining a copy of this software and
associated documentation files (the "Software"), to deal in the Software without restriction,
including without limitation the rights to use, copy, modify, merge, publish, distribute,
sublicense, and/or sell copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all copies or
substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT
NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,
DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT
OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 **************************************************************************************************/
#define BOOST_TEST_MODULE cuda_module
#include <boost/test/unit_test.hpp>
#include "gpu/Helper.cuh"
#include "gpu/GpuUtils.cuh"
#include "gpu/Assigs.cuh"
#include "gpu/Clauses.cuh"
#include "gpu/GpuHelpedSolver.h"
#include "gpu/GpuRunner.cuh"
#include "gpu/GpuMultiSolver.h"
#include "gpu/Reported.cuh"
#include "satUtils/SolverTypes.h"
#include "core/Solver.h"
#include "testUtils/TestHelper.cuh"
#include "gpu/ContigCopy.cuh"
#include <hip/hip_runtime.h>
#include <mtl/Vec.h>
#include <algorithm>
#include "utils/Utils.h"

#include "gpu/GpuRunner.cuh"
#include "gpu/GpuClauseSharer.h"

using namespace std;

namespace Glucose {



BOOST_AUTO_TEST_SUITE( GpuSolverTest )

__global__ void dUpdateAssigsGlobal(DValsPerId<VarUpdate> assigUpdates, DArr<DOneSolverAssigs> dOneSolverAssigs, DValsPerId<AggCorresp> dAggCorresps, DAssigAggregates aggregates) {
    dUpdateAssigs(assigUpdates, dOneSolverAssigs, dAggCorresps, aggregates);
}

void updateAssigsAsync(AssigsAndUpdates &assigsAndUpdates, GpuDims gpuDims, hipStream_t &stream) {
    dUpdateAssigsGlobal<<<gpuDims.blockCount, gpuDims.threadsPerBlock, 0, stream>>>(assigsAndUpdates.dAssigUpdates.get(), assigsAndUpdates.assigSet.dSolverAssigs.getDArr(),
        assigsAndUpdates.assigSet.aggCorresps.get(), assigsAndUpdates.assigSet.dAssigAggregates);
}

AssigsAndUpdates fillAndUpdateAssigs(HostAssigs &hostAssigs, GpuDims gpuDims, ContigCopier &cc, vec<AssigIdsPerSolver> &assigIdsPerSolver, hipStream_t &stream) {
    cc.clear(false);
    AssigsAndUpdates assigsAndUpdates = hostAssigs.fillAssigsAsync(cc, assigIdsPerSolver, stream);
    exitIfFalse(cc.tryCopyAsync(hipMemcpyHostToDevice, stream), POSITION);
    updateAssigsAsync(assigsAndUpdates, gpuDims, stream);
    exitIfError(hipStreamSynchronize(stream), POSITION);
    return assigsAndUpdates;
}


__global__ void dTestAssigs(DArr<DOneSolverAssigs> assigs, int solverId, DArr<MultiLBool> res) {
    for (int i = 0; i < res.size(); i++) {
        res[i] = assigs[solverId].multiLBools[i];
    }
}

// Test that the device can read the assigs
BOOST_AUTO_TEST_CASE(testAssigsTwoSolvers) {
    StreamPointer sp;
    GpuDims gpuDims {2, WARP_SIZE};
    ContigCopier cc; 
    HostAssigs hostAssigs(2, gpuDims);
    hostAssigs.growSolverAssigs(2);

    OneSolverAssigs& assig0 = hostAssigs.getAssigs(0);
    assig0.enterLock();
    BOOST_CHECK(assig0.isAssignmentAvailableLocked());
    assig0.setVarLocked(0, l_False);
    assig0.setVarLocked(1, l_Undef);
    assig0.assignmentDoneLocked();
    assig0.exitLock();

    OneSolverAssigs& assig1 = hostAssigs.getAssigs(1);
    assig1.enterLock();
    BOOST_CHECK(assig1.isAssignmentAvailableLocked());
    assig1.setVarLocked(0, l_Undef);
    assig1.setVarLocked(1, l_True);
    assig1.assignmentDoneLocked();
    assig1.exitLock();
    vec<AssigIdsPerSolver> assigIdsPerSolver;
    AssigsAndUpdates assigsAndUpdates = fillAndUpdateAssigs(hostAssigs, gpuDims, cc, assigIdsPerSolver, sp.get());
    CorrespArr<MultiLBool> res(2, true);

    // solver 0
    dTestAssigs<<<1, 1, 0, sp.get()>>>(assigsAndUpdates.assigSet.dSolverAssigs.getDArr(), 0, res.getDArr());
    res.copyAsync(hipMemcpyDeviceToHost, sp.get());
    exitIfError(hipStreamSynchronize(sp.get()), POSITION);
    BOOST_CHECK_EQUAL(1, res[0].isDef & 1);
    BOOST_CHECK_EQUAL(0, res[0].isTrue & 1);
    BOOST_CHECK_EQUAL(0, res[1].isDef & 1);

    // solver 1
    dTestAssigs<<<1, 1, 0, sp.get()>>>(assigsAndUpdates.assigSet.dSolverAssigs.getDArr(), 1, res.getDArr());
    res.copyAsync(hipMemcpyDeviceToHost, sp.get());
    exitIfError(hipStreamSynchronize(sp.get()), POSITION);
    BOOST_CHECK_EQUAL(0, res[0].isDef & 1);
    BOOST_CHECK_EQUAL(1, res[1].isDef & 1);
    BOOST_CHECK_EQUAL(1, res[1].isTrue & 1);
}

BOOST_AUTO_TEST_CASE(testAssigsTwoAssignments) {
    StreamPointer sp;
    GpuDims gpuDims {2, WARP_SIZE};
    HostAssigs hostAssigs(2, gpuDims);
    hostAssigs.growSolverAssigs(1);

    OneSolverAssigs& assig = hostAssigs.getAssigs(0);
    assig.enterLock();
    BOOST_CHECK(assig.isAssignmentAvailableLocked());
    assig.setVarLocked(0, l_True);
    assig.setVarLocked(1, l_False);
    assig.assignmentDoneLocked();
    assig.exitLock();
    assig.enterLock();
    BOOST_CHECK(assig.isAssignmentAvailableLocked());
    assig.setVarLocked(0, l_Undef);
    assig.assignmentDoneLocked();
    assig.exitLock();
    ContigCopier cc;
    vec<AssigIdsPerSolver> assigIdsPerSolver;
    DArr<DOneSolverAssigs> dAssigs = fillAndUpdateAssigs(hostAssigs, gpuDims, cc, assigIdsPerSolver, sp.get()).assigSet.dSolverAssigs.getDArr();
    CorrespArr<MultiLBool> res(2, true);

    dTestAssigs<<<1, 1, 0, sp.get()>>>(dAssigs, 0, res.getDArr());
    res.copyAsync(hipMemcpyDeviceToHost, sp.get());
    exitIfError(hipStreamSynchronize(sp.get()), POSITION);
    // first assignment
    BOOST_CHECK_EQUAL(1, res[0].isDef & 1);
    BOOST_CHECK_EQUAL(1, res[0].isTrue & 1);
    BOOST_CHECK_EQUAL(1, res[1].isDef & 1);
    BOOST_CHECK_EQUAL(0, res[1].isTrue & 1);
    // second assignment
    BOOST_CHECK_EQUAL(0, res[0].isDef & 2);
    // this var was set for the first assignment, so it should still be set now
    BOOST_CHECK_EQUAL(2, res[1].isDef & 2);
    BOOST_CHECK_EQUAL(0, res[1].isTrue & 2);
}

BOOST_AUTO_TEST_CASE(testManyAssignments) {
    StreamPointer sp;
    GpuDims gpuDims {2, WARP_SIZE};
    HostAssigs hostAssigs(assigCount(), gpuDims);
    hostAssigs.growSolverAssigs(1);

    OneSolverAssigs& assig = hostAssigs.getAssigs(0);
    for (int i = 0; i < assigCount(); i++) {
        assig.enterLock();
        BOOST_CHECK(assig.isAssignmentAvailableLocked());
        assig.setVarLocked(i, i % 2 == 0 ? l_True : l_False);
        assig.assignmentDoneLocked();
        assig.exitLock();
    }
    assig.enterLock();
    BOOST_CHECK(!assig.isAssignmentAvailableLocked());
    assig.exitLock();
    ContigCopier cc;
    vec<AssigIdsPerSolver> assigIdsPerSolver;
    auto assigsAndUpdates = fillAndUpdateAssigs(hostAssigs, gpuDims, cc, assigIdsPerSolver, sp.get());
    setAllAssigsToLastAsync(1, 1, assigsAndUpdates, sp.get());
    exitIfError(hipStreamSynchronize(sp.get()), POSITION);

    assig.enterLock();
    BOOST_CHECK(assig.isAssignmentAvailableLocked());
    assig.setVarLocked(7, l_Undef);
    assig.assignmentDoneLocked();
    assig.exitLock();

    DArr<DOneSolverAssigs> dAssigs = fillAndUpdateAssigs(hostAssigs, gpuDims, cc, assigIdsPerSolver, sp.get()).assigSet.dSolverAssigs.getDArr();
    CorrespArr<MultiLBool> res(assigCount(), true);

    dTestAssigs<<<1, 1, 0, sp.get()>>>(dAssigs, 0, res.getDArr());
    res.copyAsync(hipMemcpyDeviceToHost, sp.get());
    exitIfError(hipStreamSynchronize(sp.get()), POSITION);

    // Looking at many variables, but all on the first assignment
    for (int i = 0; i < assigCount(); i++) {
        if (i == 7) {
            BOOST_CHECK_EQUAL(0, res[i].isDef & 1);
        }
        else {
            BOOST_CHECK_EQUAL(1, res[i].isDef & 1);
            BOOST_CHECK_EQUAL(i % 2 == 0, res[i].isTrue & 1);
        }
    }
}

BOOST_AUTO_TEST_CASE(testAssigAggregates) {
    StreamPointer sp;
    GpuDims gpuDims {2, WARP_SIZE};
    HostAssigs hostAssigs(1, gpuDims);
    hostAssigs.growSolverAssigs(2);
    int assigsPerSolver = sizeof(Vals) * 8;
    for (int solv = 0; solv < 2; solv++) {
        OneSolverAssigs& assig = hostAssigs.getAssigs(solv);
        lbool l = solv == 0 ? l_True : l_False;
        for (int i = 0; i < assigsPerSolver; i++) {
            assig.enterLock();
            ASSERT_MSG(assig.isAssignmentAvailableLocked(), PRINT(i); PRINT(solv));
            assig.setVarLocked(0, i % 2 == 0 ? l : l_Undef);
            assig.assignmentDoneLocked();
            assig.exitLock();
        }
    }
    ContigCopier cc;
    vec<AssigIdsPerSolver> assigIdsPerSolver;
    AssigsAndUpdates assigsAndUpdates = fillAndUpdateAssigs(hostAssigs, gpuDims, cc, assigIdsPerSolver, sp.get());
    ASSERT_OP(~((Vals) 0), ==, assigsAndUpdates.assigSet.dAssigAggregates.startVals);
    HArr<MultiAgg> res(1, false);
    BOOST_CHECK_EQUAL(1, assigsAndUpdates.assigSet.dAssigAggregates.multiAggs.size());
    copyArrAsync(res, assigsAndUpdates.assigSet.dAssigAggregates.multiAggs, sp.get());
    exitIfError(hipStreamSynchronize(sp.get()), POSITION);
    BOOST_CHECK_EQUAL((((Vals) 1) << (assigsPerSolver / 2)) - 1, res[0].canBeTrue);
    BOOST_CHECK_EQUAL(((((Vals) 1) << (assigsPerSolver / 2)) - 1) << (assigsPerSolver / 2), res[0].canBeFalse);
    BOOST_CHECK_EQUAL(~((Vals) 0), res[0].canBeUndef);
}

__global__ void dTestCopyClause(DClauses clauses, DArr<Lit> res) {
    int p = getStartPosForClause(res.size(), 0);
    for (int i = 0; i < res.size(); i++) {
        res[i] = clauses.get(res.size(), p + WARP_SIZE * i);
    }
}

// Tests adding clauses, and that the device can read them
BOOST_AUTO_TEST_CASE(testAddClauseHost) {
    StreamPointer sp;
    CorrespArr<int> clausesCountPerThread(2, true);
    GpuDims gpuDims(2, WARP_SIZE);
    vec<unsigned long> globalStats(100, 0);
    HostClauses hClauses(gpuDims, 0.99, false, globalStats);
    addClause(hClauses, {mkLit(4), mkLit(2)});
    CorrespArr<Lit> cra(2, false);

    copyToDeviceAsync(hClauses, sp.get(), gpuDims);

    ContigCopier cc;
    RunInfo runInfo = hClauses.makeRunInfo(sp.get(), cc);
    cc.tryCopyAsync(hipMemcpyHostToDevice, sp.get());
    dTestCopyClause<<<1, 1, 0, sp.get()>>>(runInfo.getDClauses(), cra.getDArr());
    cra.copyAsync(hipMemcpyDeviceToHost, sp.get());
    exitIfError(hipStreamSynchronize(sp.get()), POSITION);
    BOOST_CHECK_EQUAL(mkLit(4).x, cra[0].x);
    BOOST_CHECK_EQUAL(mkLit(2).x, cra[1].x);
}

__global__ void dTestReporter(DReporter<ReportedClause> dreporter) {
    // reports the clause id 0 of size 1 to the assignment 0 and solver 0
    dreporter.report(ReportedClause{1, 0, GpuCref{1, 0}}, getThreadId());
}

template<typename T> __global__ void dClear(DReporter<T> rep) {
    rep.clear();
}

// Tests that the gpu reporter can report wrong clauses and that the cpu can read them
BOOST_AUTO_TEST_CASE(testReported) {
    GpuOptions ops;
    setDefaultOptions(ops);
    GpuFixture fx(ops, 3, 10);
    hipStream_t &stream = fx.gpuClauseSharer.sp.get();
    
    fx.addClause({mkLit(3)});
    copyToDeviceAsync(*fx.gpuClauseSharer.clauses, stream, GpuDims {32, 1});
    BOOST_CHECK_EQUAL(1, fx.gpuClauseSharer.clauses->getClauseCount(1));

    int solvId = 0;

    AssigIdsPerSolver aips;
    aips.startAssigId = 0;
    aips.assigCount = 1;
    vec<AssigIdsPerSolver> assigIds(1, aips);

    ContigCopier gpuToCpuCc;
    Reporter<ReportedClause> reporter(gpuToCpuCc, stream, 4, 4);

    auto dReporter = reporter.getDReporter();
    dClear<<<1, 1, 0, stream>>>(dReporter);
    dTestReporter<<< 1, 1, 0, stream>>>(dReporter);

    exitIfFalse(gpuToCpuCc.tryCopyAsync(hipMemcpyDeviceToHost, stream), POSITION);
    exitIfError(hipStreamSynchronize(stream), POSITION);
   
    vec<ReportedClause> wcl; 
    reporter.getCopiedToHost(wcl);

    fx.gpuClauseSharer.reported->fill(assigIds, wcl);

    MinHArr<Lit> lits;
    GpuClauseId gpuClauseId;
    BOOST_CHECK(fx.gpuClauseSharer.reported->popReportedClause(solvId, lits, gpuClauseId));
    BOOST_CHECK_EQUAL(1, lits.size());
    BOOST_CHECK(lits[0] == mkLit(3));
    BOOST_CHECK_EQUAL(0, gpuClauseId);

    BOOST_CHECK(!fx.gpuClauseSharer.reported->popReportedClause(solvId, lits, gpuClauseId));

    exitIfLastError(POSITION);
}

int getReportedClausesCount(Reported &reported, int solverId) {
    MinHArr<Lit> lits;
    GpuClauseId gpuClauseId;
    int count = 0;
    while(reported.popReportedClause(solverId, lits, gpuClauseId)) count++;
    return count;
}

GpuDims getGpuDims(GpuOptions opts) { 
    return GpuDims {opts.blockCount, opts.threadsPerBlock};
}

// There's not a full solver in this test, but everything else
BOOST_AUTO_TEST_CASE(testClausesAssigsReported) {
    GpuOptions ops;
    setDefaultOptions(ops);
    GpuFixture fx(ops, 3, 3);

    addClause(*fx.gpuClauseSharer.clauses, {mkLit(0)});
    addClause(*fx.gpuClauseSharer.clauses, {mkLit(1)});
    addClause(*fx.gpuClauseSharer.clauses, {mkLit(2)});
    copyToDeviceAsync(*fx.gpuClauseSharer.clauses, fx.gpuClauseSharer.sp.get(), getGpuDims(ops));

    // assigs for solver 0
    OneSolverAssigs& assig0 = fx.gpuClauseSharer.assigs->getAssigs(0);
    assig0.enterLock();
    assig0.setVarLocked(0, l_False);
    assig0.setVarLocked(1, l_True);
    assig0.setVarLocked(2, l_True);
    assig0.assignmentDoneLocked();
    assig0.exitLock();

    assig0.enterLock();
    assig0.setVarLocked(0, l_True);
    assig0.setVarLocked(1, l_False);
    assig0.setVarLocked(2, l_False);
    assig0.assignmentDoneLocked();
    assig0.exitLock();

    // assigs for solv 1
    OneSolverAssigs& assig1 = fx.gpuClauseSharer.assigs->getAssigs(1);
    assig1.enterLock();
    assig1.setVarLocked(0, l_True);
    assig1.setVarLocked(1, l_False);
    assig1.setVarLocked(2, l_True);
    assig1.assignmentDoneLocked();
    assig1.exitLock();
    execute(fx.gpuClauseSharer);

    BOOST_CHECK_EQUAL(3, getReportedClausesCount(*fx.gpuClauseSharer.reported, 0));
    BOOST_CHECK_EQUAL(1, getReportedClausesCount(*fx.gpuClauseSharer.reported, 1));
    BOOST_CHECK_EQUAL(0, getReportedClausesCount(*fx.gpuClauseSharer.reported, 2));

    assig0.enterLock();
    assig0.setVarLocked(1, l_True);
    assig0.assignmentDoneLocked();
    assig0.exitLock();
    execute(fx.gpuClauseSharer);
    BOOST_CHECK_EQUAL(1, getReportedClausesCount(*fx.gpuClauseSharer.reported, 0));

    BOOST_CHECK_EQUAL(0, getReportedClausesCount(*fx.gpuClauseSharer.reported, 1));
}

// Test that the gpu can read assigs and report the appropriate wrong clause
BOOST_AUTO_TEST_CASE(testFindClausesMultiThread) {
    GpuOptions ops;
    setDefaultOptions(ops);
    ops.blockCount = 1;
    ops.threadsPerBlock = 32;
    GpuDims gpuDims {ops.blockCount, ops.threadsPerBlock};
    GpuFixture fx(ops, 3, 1);
    hipStream_t &stream = fx.gpuClauseSharer.sp.get();

    OneSolverAssigs &oneSolverAssigs = fx.gpuClauseSharer.assigs->getAssigs(0);
    oneSolverAssigs.enterLock();
    oneSolverAssigs.setVarLocked(0, l_False);
    oneSolverAssigs.setVarLocked(1, l_True);
    oneSolverAssigs.setVarLocked(2, l_Undef);
    oneSolverAssigs.assignmentDoneLocked();
    oneSolverAssigs.exitLock();

    // test copying the clauses several times
    addClause(*fx.gpuClauseSharer.clauses, {mkLit(0), mkLit(1)});
    copyToDeviceAsync(*fx.gpuClauseSharer.clauses, stream, gpuDims);
    addClause(*fx.gpuClauseSharer.clauses, {mkLit(0), ~mkLit(1)});
    copyToDeviceAsync(*fx.gpuClauseSharer.clauses, stream, gpuDims);
    addClause(*fx.gpuClauseSharer.clauses, {~mkLit(1), mkLit(2)});
    copyToDeviceAsync(*fx.gpuClauseSharer.clauses, stream, gpuDims);

    ContigCopier cc;
    std::unique_ptr<AssigsAndUpdates> assigsAndUpdates;
    std::unique_ptr<Reporter<ReportedClause>> reporter;

    vec<AssigIdsPerSolver> assigIdsPerSolver;
    execute(fx.gpuClauseSharer);

    int *lits1, *lits2; 
    int count1, count2; 
    long gpuClauseId;
    BOOST_CHECK(fx.gpuClauseSharer.popReportedClause(0, lits1, count1, gpuClauseId));
    BOOST_CHECK(fx.gpuClauseSharer.popReportedClause(0, lits2, count2, gpuClauseId));

    MinHArr<Lit> cl1 {(size_t) count1, (Lit*) lits1};
    MinHArr<Lit> cl2 {(size_t) count2, (Lit*) lits2};

    MinHArr<Lit> forCl2, forCl3;
    if (cl1[0] == mkLit(0)) {
        forCl2 = cl1;
        forCl3 = cl2;
    } else {
        forCl2 = cl2;
        forCl3 = cl1;
    }

    BOOST_CHECK_EQUAL(2, forCl2.size());
    BOOST_CHECK_EQUAL(mkLit(0).x, forCl2[0].x);
    BOOST_CHECK_EQUAL((~mkLit(1)).x, forCl2[1].x);

    BOOST_CHECK_EQUAL(2, forCl3.size());
    BOOST_CHECK_EQUAL((~mkLit(1)).x, forCl3[0].x);
    BOOST_CHECK_EQUAL(mkLit(2).x, forCl3[1].x);

    BOOST_CHECK(!fx.gpuClauseSharer.popReportedClause(0, lits1, count1, gpuClauseId));
}

BOOST_AUTO_TEST_CASE(SolverImportBinary) {
    GpuOptions ops;
    setDefaultOptions(ops);
    GpuFixture fx(ops, 3, 1);
    GpuHelpedSolver& solver = *(fx.solvers[0]);
    // level 0: var 0 is true
    solver.uncheckedEnqueue(mkLit(0));
    solver.newDecisionLevel();
    // level 1: var 2 is true
    solver.uncheckedEnqueue(mkLit(2));
    BOOST_CHECK((l_True == solver.value(2)));

    // add gpu clause: 0 implies 1
    addClause(*fx.gpuClauseSharer.clauses, {~mkLit(0), mkLit(1)});
    fx.executeAndImportClauses();

    solver.propagate();
    BOOST_CHECK((l_True == solver.value(0)));
    BOOST_CHECK((l_True == solver.value(1)));
    // 2 should have been unset
    BOOST_CHECK((l_Undef == solver.value(2)));

    BOOST_CHECK((0 == solver.level(0)));
    BOOST_CHECK((0 == solver.level(1)));
    fx.checkReportedImported(1, 0, false);
}

// The reason to have this test is that it would have failed if it had been there
// The only assig aggregate bits set to the last value were those which were
// used. It needs to be all of them. This test is for this case
BOOST_AUTO_TEST_CASE(testOneAssignmentThenTwo) {
    GpuOptions ops;
    setDefaultOptions(ops);
    GpuFixture fx(ops, 4, 1);
    hipStream_t &stream = fx.gpuClauseSharer.sp.get();
    GpuHelpedSolver& solver = *(fx.solvers[0]);
    addClause(*fx.gpuClauseSharer.clauses, {~mkLit(0), ~mkLit(1), mkLit(2)});
    copyToDeviceAsync(*fx.gpuClauseSharer.clauses, stream, getGpuDims(ops));

    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(0));
    solver.tryCopyTrailForGpu(solver.decisionLevel());
    execute(fx.gpuClauseSharer);

    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(3));
    solver.tryCopyTrailForGpu(solver.decisionLevel());
    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(1));
    solver.tryCopyTrailForGpu(solver.decisionLevel());
    execute(fx.gpuClauseSharer);
    BOOST_CHECK_EQUAL(1, getReportedClausesCount(*fx.gpuClauseSharer.reported, 0));
}


BOOST_AUTO_TEST_CASE(SolverDoesntImportSameClauseTwice) {
    GpuOptions ops;
    setDefaultOptions(ops);
    GpuFixture fx(ops, 3, 1);
    GpuHelpedSolver& solver = *(fx.solvers[0]);

    // Solver will copy two assignments to gpu, each one will get back the same clause
    // test that the solver only imports the clause once
    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(0));
    solver.tryCopyTrailForGpu(1);

    solver.cancelUntil(0);
    solver.newDecisionLevel();
    solver.uncheckedEnqueue(~mkLit(1));
    solver.tryCopyTrailForGpu(1);
    addClause(*fx.gpuClauseSharer.clauses, {~mkLit(0), mkLit(1)});
    // no need to call copyToDevice because execute does it
    execute(fx.gpuClauseSharer);

    bool foundEmptyClause = false;

    solver.gpuImportClauses(foundEmptyClause);

    BOOST_CHECK_EQUAL(1, fx.gpuClauseSharer.getOneSolverStat(0, reportedClauses));
}

// This test is about the same as the previous one, except that we start a gpu run before the second assignment is sent
// So clauses for both assignment will be reported in distinct clause batches
BOOST_AUTO_TEST_CASE(SolverDoesntImportSameClauseTwiceOnSuccessiveGpuExecutions) {
    GpuOptions ops;
    setDefaultOptions(ops);
    GpuFixture fx(ops, 3, 1);
    GpuHelpedSolver& solver = *(fx.solvers[0]);
    addClause(*fx.gpuClauseSharer.clauses, {~mkLit(0), mkLit(1)});

    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(0));
    solver.tryCopyTrailForGpu(1);
    fx.gpuClauseSharer.gpuRun();

    solver.cancelUntil(0);
    solver.newDecisionLevel();
    solver.uncheckedEnqueue(~mkLit(1));
    solver.tryCopyTrailForGpu(1);
    fx.gpuClauseSharer.gpuRun();
    bool foundEmptyClause;
    solver.gpuImportClauses(foundEmptyClause);
    fx.gpuClauseSharer.gpuRun();
    solver.gpuImportClauses(foundEmptyClause);

    BOOST_CHECK_EQUAL(1, fx.gpuClauseSharer.getOneSolverStat(0, reportedClauses));
}

// Test that if a clause has been imported and then deleted, it can be imported again
BOOST_AUTO_TEST_CASE(SolverCanReimportClause) {
    // In this test: we add two clauses on the gpu, both get imported, then we reduceDb
    // on the cpu, so one gets deleted, and we test that we can import it again
    GpuOptions ops;
    setDefaultOptions(ops);
    GpuFixture fx(ops, 5, 1);
    GpuHelpedSolver& solver = *(fx.solvers[0]);

    // reason for having clauses of size 3: if they were of size 2, they'd be permanently learned so we couldn't test anything
    addClause(*fx.gpuClauseSharer.clauses, {~mkLit(0), ~mkLit(1), mkLit(4)});
    addClause(*fx.gpuClauseSharer.clauses, {~mkLit(0), ~mkLit(1), mkLit(3)});

    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(0));
    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(1));
    BOOST_CHECK_EQUAL(0, (int) solver.stats[learnedFromGpu]);
    BOOST_CHECK_EQUAL(0, (int) solver.stats[nbImported]);
    fx.executeAndImportClauses();

    solver.cancelUntil(0);
    BOOST_CHECK_EQUAL(2, (int) solver.stats[learnedFromGpu]);
    solver.reduceDB();
    BOOST_CHECK_EQUAL(1, (int) solver.stats[learnedFromGpu]);
    BOOST_CHECK_EQUAL(2, (int) solver.stats[nbImported]);

    solver.cancelUntil(0);
    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(0));
    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(1));
    // just so that the first clause doesn't need to be imported by the gpu
    solver.propagate();
    fx.executeAndImportClauses();
    BOOST_CHECK_EQUAL(3, (int) solver.stats[nbImported]);
    // the second clause that was removed during the reduceDB has been re-added
    BOOST_CHECK_EQUAL(2, (int) solver.stats[learnedFromGpu]);
}

BOOST_AUTO_TEST_CASE(TwoSolverImportBinary) {
    GpuOptions ops;
    setDefaultOptions(ops);
    GpuFixture fx(ops, 3, 2);

    fx.solvers[0]->uncheckedEnqueue(mkLit(0));
    fx.solvers[1]->uncheckedEnqueue(~mkLit(0));

    fx.solvers[0]->propagate();
    fx.solvers[1]->propagate();

    BOOST_CHECK((l_Undef == fx.solvers[0]->value(1)));
    BOOST_CHECK((l_Undef == fx.solvers[1]->value(1)));

    addClause(*fx.gpuClauseSharer.clauses, {~mkLit(0), mkLit(1)});
    addClause(*fx.gpuClauseSharer.clauses, {mkLit(0), ~mkLit(1)});

    vec<CRef> v;
    fx.executeAndImportClauses(v);

    fx.solvers[0]->propagate();
    fx.solvers[1]->propagate();
    BOOST_CHECK_EQUAL(1, fx.solvers[0]->stats[nbImported]);
    BOOST_CHECK_EQUAL(1, fx.solvers[1]->stats[nbImported]);
    BOOST_CHECK((l_True == fx.solvers[0]->value(1)));
    BOOST_CHECK((l_False == fx.solvers[1]->value(1)));
}

// Test that assigs on the gpu do get unset
BOOST_AUTO_TEST_CASE(SolverUnsets) {
    GpuOptions ops;
    setDefaultOptions(ops);
    GpuFixture fx(ops, 2, 1);
    // addClause(*fx.gpuClauseSharer.clauses, {mkLit(0), mkLit(1)});
    GpuHelpedSolver& solver = *fx.solvers[0];
    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(1));
    vec<CRef> v;
    fx.executeAndImportClauses(v);
    fx.checkReportedImported(0, 0, false);

    solver.cancelUntil(0);
    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(0));

    addClause(*fx.gpuClauseSharer.clauses, {~mkLit(0), mkLit(1)});
    fx.executeAndImportClauses(v);
    // the clause just added should have been imported because 1 is unset
    fx.checkReportedImported(1, 0, false);
}

BOOST_AUTO_TEST_CASE(SolverImportUnary) {
    GpuOptions ops;
    setDefaultOptions(ops);
    GpuFixture fx(ops, 2, 1);
    GpuHelpedSolver& solver = *(fx.solvers[0]);
    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(1));

    BOOST_CHECK((l_True == solver.value(1)));

    addClause(*fx.gpuClauseSharer.clauses, {mkLit(0)});
    fx.executeAndImportClauses();
    BOOST_CHECK_EQUAL(1, solver.stats[nbImportedUnit]);
    BOOST_CHECK_EQUAL(1, solver.stats[nbImported]);
    solver.propagate();

    BOOST_CHECK((l_True == solver.value(0)));
    BOOST_CHECK((l_Undef == solver.value(1)));

    BOOST_CHECK((0 == solver.level(0)));
}

BOOST_AUTO_TEST_CASE(SolverHasManyClausesReported) {
    GpuOptions ops;
    setDefaultOptions(ops);
    int varCount = 20;
    GpuFixture fx(ops, varCount, 1);
    GpuHelpedSolver& solver = *(fx.solvers[0]);


    for (int i = 0; i < varCount; i++) {
        // gpu clause: var is true
        addClause(*fx.gpuClauseSharer.clauses, {mkLit(i)});
        // solver learns clause
        fx.executeAndImportClauses();
        BOOST_CHECK((l_True == solver.value(i)));
    }
    fx.checkReportedImported(varCount, 0, true);
}

BOOST_AUTO_TEST_CASE(SolverHasManyClausesReportedAllAtOnce) {
    GpuOptions ops;
    setDefaultOptions(ops);
    ops.blockCount = 2;
    ops.threadsPerBlock = 32;
    // The point of having that many is to have more than one clause per thread
    int varCount = 4000;
    GpuFixture fx(ops, varCount, 1, 5000);
    GpuHelpedSolver& solver = *(fx.solvers[0]);

    for (int i = 0; i < varCount; i++) {
        // a third of the variables are already true, so the clauses won't be imported
        if (i % 3 == 0) {
            solver.newDecisionLevel();
            solver.uncheckedEnqueue(mkLit(i));
        }
        addClause(*fx.gpuClauseSharer.clauses, {mkLit(i)});
    }
    // solver learns clause
    fx.executeAndImportClauses();
    for (int i = 0; i < varCount; i++) {
        // Those were set at the beginning, so their clauses haven't been imported, and they've
        // been unset because of the other literals added, so they're not set any more
        if (i % 3 == 0) BOOST_CHECK((l_Undef == solver.value(i)));
        else ASSERT_OP_MSG(l_True, ==, solver.value(i), PRINT(i));
    }
    fx.checkReportedImported(varCount - getRequired(varCount, 3), 0, true);
}


BOOST_AUTO_TEST_CASE(OneInstanceTwoAssignments) {
    // in this test: we test that if some clauses become useful at some point,
    // then they will be imported.
    GpuOptions ops;
    setDefaultOptions(ops);
    GpuFixture fx(ops, 4, 1);

    GpuHelpedSolver& solver = *(fx.solvers[0]);

    addClause(*fx.gpuClauseSharer.clauses, {~mkLit(0), mkLit(2)});
    addClause(*fx.gpuClauseSharer.clauses, {~mkLit(1), mkLit(3)});

    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(0));

    solver.tryCopyTrailForGpu(solver.decisionLevel());
    solver.cancelUntil(0);
    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(1));
    solver.tryCopyTrailForGpu(solver.decisionLevel());
    solver.cancelUntil(0);
    BOOST_CHECK_EQUAL(0, fx.gpuClauseSharer.getOneSolverStat(0, reportedClauses));
    fx.executeAndImportClauses();

    // at this point, both clauses should have been imported
    // check that when propagating, these clauses are actually used
    BOOST_CHECK_EQUAL(2, fx.gpuClauseSharer.getOneSolverStat(0, reportedClauses));
    BOOST_CHECK_EQUAL(2, solver.stats[nbImported]);
    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(0));
    solver.propagate();
    BOOST_CHECK((l_True == solver.value(2)));

    solver.cancelUntil(0);
    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(1));
    solver.propagate();
    BOOST_CHECK((l_True == solver.value(3)));
}

BOOST_AUTO_TEST_CASE(SolverClauseKeptAfterImport) {
    GpuOptions ops;
    setDefaultOptions(ops);
    GpuFixture fx(ops, 3, 1);
    GpuHelpedSolver& solver = *(fx.solvers[0]);

    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(0));

    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(1));

    addClause(*fx.gpuClauseSharer.clauses, {~mkLit(0), ~mkLit(1), mkLit(2)});
    fx.executeAndImportClauses();

    BOOST_CHECK((l_True == solver.value(0)));
    BOOST_CHECK((l_True == solver.value(1)));
    BOOST_CHECK((l_True == solver.value(2)));

    solver.cancelUntil(1); // should cancel 1 and 2
    BOOST_CHECK((l_True == solver.value(0)));
    BOOST_CHECK((l_Undef == solver.value(1)));
    BOOST_CHECK((l_Undef == solver.value(2)));

    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(1));
    solver.propagate();
    BOOST_CHECK((l_True == solver.value(0)));
    BOOST_CHECK((l_True == solver.value(1)));
    BOOST_CHECK((l_True == solver.value(2)));

    fx.checkReportedImported(1, 0, false);
}

BOOST_AUTO_TEST_CASE(SolverImportFalseClauseDifferentLevel) {
    GpuOptions ops;
    setDefaultOptions(ops);
    GpuFixture fx(ops, 3, 1);
    GpuHelpedSolver& solver = *(fx.solvers[0]);

    solver.addClause(~mkLit(1), mkLit(2));

    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(0));
    solver.propagate();

    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(1));

    solver.propagate();
    BOOST_CHECK((l_True == solver.value(0)));
    BOOST_CHECK((l_True == solver.value(1)));
    BOOST_CHECK((l_True == solver.value(2)));

    BOOST_CHECK((1 == solver.level(0)));
    BOOST_CHECK((2 == solver.level(1)));
    BOOST_CHECK((2 == solver.level(2)));

    // at this point, 0 and 1 have different levels. So 1 should now just be implied by 0
    addClause(*fx.gpuClauseSharer.clauses, {~mkLit(0), ~mkLit(1)});
    fx.executeAndImportClauses();

    BOOST_CHECK((l_True == solver.value(0)));
    BOOST_CHECK((l_False == solver.value(1)));
    BOOST_CHECK((l_Undef == solver.value(2)));

    BOOST_CHECK((1 == solver.level(0)));
    BOOST_CHECK((1 == solver.level(1)));

    fx.checkReportedImported(1, 0, false);
}

BOOST_AUTO_TEST_CASE(testDeduceEmptyClause) {
    GpuOptions ops;
    setDefaultOptions(ops);
    GpuFixture fx(ops, 1, 1);
    GpuHelpedSolver& solver = *(fx.solvers[0]);

    solver.uncheckedEnqueue(mkLit(0));
    solver.propagate();
    BOOST_CHECK((l_True == solver.value(0)));

    addClause(*fx.gpuClauseSharer.clauses, {~mkLit(0)});

    fx.execute();
    exitIfLastError(POSITION);
    // we have the clauses v(0) == true and v(0) == false
    BOOST_CHECK((l_False == solver.solve()));
    exitIfLastError(POSITION);
}

BOOST_AUTO_TEST_CASE(findConflict) {
    GpuOptions ops;
    setDefaultOptions(ops);
    GpuFixture fx(ops, 2, 1);
    GpuHelpedSolver& solver = *(fx.solvers[0]);

    solver.addClause(~mkLit(0), mkLit(1));
    solver.addClause(mkLit(0), ~mkLit(1));
    solver.newDecisionLevel();

    solver.uncheckedEnqueue(mkLit(0));
    solver.propagate();

    BOOST_CHECK_EQUAL(1, solver.level(0));
    BOOST_CHECK_EQUAL(1, solver.level(1));

    addClause(*fx.gpuClauseSharer.clauses, {~mkLit(0), ~mkLit(1)});
    // we don't call executeAndImportClauses because the import has to be
    // done during solve()
    fx.execute();
    BOOST_CHECK_EQUAL(1, fx.gpuClauseSharer.getGlobalStat(gpuClauses));

    BOOST_CHECK(l_True == solver.solve());
    BOOST_CHECK_EQUAL(1, solver.stats[nbImported]);

    BOOST_CHECK(l_False == solver.modelValue(0));
    BOOST_CHECK(l_False == solver.modelValue(1));

    exitIfLastError(POSITION);
}

BOOST_AUTO_TEST_CASE(testReduceDb) {
    GpuOptions ops;
    setDefaultOptions(ops);
    GpuFixture fx(ops, 5, 1);
    hipStream_t &stream = fx.gpuClauseSharer.sp.get();

    addClause(*fx.gpuClauseSharer.clauses, {~mkLit(1), mkLit(3), mkLit(4)});
    // Adding this clause (the one which will be used) last so that it will have to
    // be passed from one thread to another
    addClause(*fx.gpuClauseSharer.clauses, {~mkLit(0), mkLit(2), mkLit(4)});

    copyToDeviceAsync(*fx.gpuClauseSharer.clauses, stream, getGpuDims(ops));
    BOOST_CHECK_EQUAL(2, fx.gpuClauseSharer.getGlobalStat(gpuClauses));
    BOOST_CHECK_EQUAL(6, fx.gpuClauseSharer.getGlobalStat(gpuClauseLengthSum));

    GpuHelpedSolver& solver = *(fx.solvers[0]);
    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(0));
    solver.newDecisionLevel();
    solver.uncheckedEnqueue(~mkLit(4));

    vec<CRef> ignored;
    fx.executeAndImportClauses(ignored);
    BOOST_CHECK(l_True == solver.value(2));
    BOOST_CHECK(l_Undef == solver.value(3));

    printf("reduce db\n");
    fx.gpuClauseSharer.clauses->reduceDb(stream);

    BOOST_CHECK_EQUAL(1, fx.gpuClauseSharer.getGlobalStat(gpuClauses));
    BOOST_CHECK_EQUAL(3, fx.gpuClauseSharer.getGlobalStat(gpuClauseLengthSum));

    // the second clause should have been removed because it wasn't used before
    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(1));
    fx.executeAndImportClauses(ignored);
    BOOST_CHECK(l_Undef == solver.value(3));

    int rep = fx.gpuClauseSharer.getOneSolverStat(0, reportedClauses);
    // now check that adding more clauses works fine after the reduce db
    addClause(*fx.gpuClauseSharer.clauses, {~mkLit(0), ~mkLit(3)});
    fx.executeAndImportClauses(ignored);

    BOOST_CHECK_EQUAL(rep + 1, fx.gpuClauseSharer.getOneSolverStat(0, reportedClauses));

    BOOST_CHECK(l_False == solver.value(3));
}

BOOST_AUTO_TEST_CASE(testMods) {
    BOOST_CHECK_EQUAL(3, getLargestSameMod(3, 4, 4));
    BOOST_CHECK_EQUAL(3, getLargestSameMod(3, 5, 4));
    BOOST_CHECK_EQUAL(3, getLargestSameMod(3, 6, 4));
    BOOST_CHECK_EQUAL(7, getLargestSameMod(3, 7, 4));

    BOOST_CHECK_EQUAL(0, getRequired(0, 4));
    BOOST_CHECK_EQUAL(1, getRequired(1, 4));
    BOOST_CHECK_EQUAL(1, getRequired(4, 4));
    BOOST_CHECK_EQUAL(2, getRequired(5, 4));
}

BOOST_AUTO_TEST_CASE(testSolverPassesManyAssignments) {
    GpuOptions ops;
    setDefaultOptions(ops);
    GpuFixture fx(ops, 64, 3, 100);

    GpuHelpedSolver& solver = *(fx.solvers[0]);
    for (int i = 0; i < 32; i++) {
        addClause(*fx.gpuClauseSharer.clauses, {~mkLit(2 * i), mkLit(2 * i + 1)});
        solver.cancelUntil(0);
        solver.newDecisionLevel();
        solver.uncheckedEnqueue(mkLit(2 * i));
        solver.tryCopyTrailForGpu(solver.decisionLevel());
    }
    execute(fx.gpuClauseSharer);

    bool foundEmptyClause = false;
    solver.gpuImportClauses(foundEmptyClause);

    BOOST_CHECK_EQUAL(0, solver.stats[nbFailureFindAssignment]);
    BOOST_CHECK_EQUAL(1, solver.stats[nbImportedValid]);
    BOOST_CHECK_EQUAL(32, solver.stats[nbImported]);

    // check the clauses have really been learned and can be propagated
    solver.cancelUntil(0);
    for (int i = 0; i < 32; i++) {
        solver.newDecisionLevel();
        solver.uncheckedEnqueue(mkLit(2 * i));
        BOOST_CHECK(l_Undef == solver.value(2 * i + 1));
        solver.propagate();
        BOOST_CHECK(l_True == solver.value(2 * i + 1));
    }
}

BOOST_AUTO_TEST_CASE(testGpuMultiSolver) {
    GpuOptions ops;
    setDefaultOptions(ops);
    CommonOptions commonOpts;
    Finisher finisher;
    CompositionRoot co(ops, commonOpts, finisher, 2);

    GpuMultiSolver &msolver = *co.gpuMultiSolver;

    msolver.addClause({mkLit(0)});
    msolver.addClause({~mkLit(0), mkLit(1)});
    BOOST_CHECK((l_True == msolver.solve(1)));
}

BOOST_AUTO_TEST_CASE(testSendClauseToGpu) {
    GpuOptions ops;
    setDefaultOptions(ops);
    GpuFixture fx(ops, 3, 1);
    GpuHelpedSolver& solver = *(fx.solvers[0]);

    solver.addClause(~mkLit(0), mkLit(1));
    solver.addClause(~mkLit(0), ~mkLit(1));
    solver.newDecisionLevel();
    solver.uncheckedEnqueue(mkLit(0));

    BOOST_CHECK_EQUAL(0, solver.conflicts);
    BOOST_CHECK_EQUAL(0, solver.stats[propagations]);

    bool b1;
    vec<Lit> learned_clause, selectors;
    bool blocked = false;
    solver.propagateAndMaybeLearnFromConflict(b1, blocked, learned_clause, selectors);
    BOOST_CHECK_EQUAL(1, solver.conflicts);
    BOOST_CHECK_EQUAL(1, solver.stats[propagations]);
    execute(fx.gpuClauseSharer);

    solver.gpuImportClauses(b1);
    // if already present:
    // The gpu clauses don't need to learn the clause ~mkLit(0) that the solver has just found because the assignment mkLit(0) was
    // already know to be not useful (because the gpu clauses already have the clause ~mkLit(0)
    copyToDeviceAsync(*fx.gpuClauseSharer.clauses, fx.gpuClauseSharer.sp.get(), getGpuDims(ops));
    BOOST_CHECK_EQUAL(1, fx.gpuClauseSharer.getGlobalStat(gpuClauses));
    BOOST_CHECK_EQUAL(1, fx.gpuClauseSharer.getGlobalStat(gpuClauseLengthSum));
}

BOOST_AUTO_TEST_CASE(testClauseBatch) {
    ClauseBatch clBatch;
    clBatch.addClause(23);
    clBatch.addLit(mkLit(1));
    clBatch.addLit(mkLit(2));

    MinHArr<Lit> minHArr;
    GpuClauseId gpuClauseId;
    BOOST_CHECK((clBatch.popClause(minHArr, gpuClauseId)));
    BOOST_CHECK_EQUAL(23, gpuClauseId);
    BOOST_CHECK_EQUAL(2, minHArr.size());
    BOOST_CHECK_EQUAL(toInt(mkLit(1)), toInt(minHArr[0]));
    BOOST_CHECK_EQUAL(toInt(mkLit(2)), toInt(minHArr[1]));
}


BOOST_AUTO_TEST_SUITE_END()

}
